// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// Thread block size
#define BLOCK_SIZE 16
// Matrix dimensions
#define WA 4 // Matrix A width
#define HA 4 // Matrix A height
#define HB WA  // Matrix B height
//#define WC WB  // Matrix C width 
#define HC HA // Matrix C height

// includes, project
#include <cutil.h>

// includes, kernels
#include "matrixMul_kernel.cu"
////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    runTest(argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv){
    CUT_DEVICE_INIT();

    // set seed for rand()
    srand(2006);
	
	int WB = 10000;
	int WC = WB;											// WB = WC = 10000 PRA FUNCIONAR
    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);				//CALLOC ?
    unsigned int size_B = WB * HB;								
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    // initialize host memory								// USAR NOSSAS FUNÇÕES DE DE PREENCHIMENTO
    randomInit(h_A, size_A);		
    randomInit(h_B, size_B);

    // allocate device memory
    float* d_A;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_A, mem_size_A));
    float* d_B;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_B, mem_size_B));

    // copy host memory to device
    CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);
    
    // create and start timer
    unsigned int timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x);						// HC / threads  = 0 talvez HC % threads ? 	grid unidimencional

    // execute the kernel
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);			// só coluna é passada

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // copy result from device to host
    CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );

    // stop and destroy timer
    CUT_SAFE_CALL(cutStopTimer(timer));
    printf("Processing time: %f (ms) \n", cutGetTimerValue(timer));
    CUT_SAFE_CALL(cutDeleteTimer(timer));

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    CUDA_SAFE_CALL(hipFree(d_A));
    CUDA_SAFE_CALL(hipFree(d_B));
    CUDA_SAFE_CALL(hipFree(d_C));
}
